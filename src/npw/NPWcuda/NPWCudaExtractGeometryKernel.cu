#include "hip/hip_runtime.h"
/*
 * NPWCudaExtractGeometryKernel.cu
 *
 *  Created on: 19/07/2010
 *      Author: bro86j
 */

#include <stdio.h>

#include "NPWCudaGeometryFunctions.h"
#include "NPWCudaShapeDetermination.h"

#include "NPWCudaDataPointer.h"


/*
 * extracts geometry data from two images
 */
__global__ void ExtractGeometryFromImagesKernel(
        float *      d_geometryData,
        float *      d_image1Data,
        float *      d_image2Data,
        unsigned int imageDimX,
        unsigned int imageDimY,
        unsigned int imageDimZ,
        float        innerWeight,
        float        outerWeight,
        float        image1Min,
        float        image2Min,
        float        invBinSize1,
        float        invBinSize2
        )
{
    // these are neighborhood indices,
    // corresponding to the first voxel of the neighborhood
    unsigned int x = blockIdx.x;
    unsigned int y = blockIdx.y;
    unsigned int z = threadIdx.x;

    // copy neighborhood to local memory, to speed things up
    float2 neighborhood_cache[8];
    for ( int i = 0; i < 8; ++i )
    {
        neighborhood_cache[i] = make_float2(0.0f,0.0f);
    }

    // check if this thread neighborhood is within
    if ( x < imageDimX-1 && y < imageDimY-1 && z < imageDimZ-1 )
    {
        // get 5 tets
        neighborhood_cache[0].x = d_image1Data[ getArrayIndex3D(x  ,y  ,z  ,imageDimX,imageDimY) ];
        neighborhood_cache[0].y = d_image2Data[ getArrayIndex3D(x  ,y  ,z  ,imageDimX,imageDimY) ];

        neighborhood_cache[1].x = d_image1Data[ getArrayIndex3D(x+1,y  ,z  ,imageDimX,imageDimY) ];
        neighborhood_cache[1].y = d_image2Data[ getArrayIndex3D(x+1,y  ,z  ,imageDimX,imageDimY) ];

        neighborhood_cache[2].x = d_image1Data[ getArrayIndex3D(x  ,y+1,z  ,imageDimX,imageDimY) ];
        neighborhood_cache[2].y = d_image2Data[ getArrayIndex3D(x  ,y+1,z  ,imageDimX,imageDimY) ];

        neighborhood_cache[3].x = d_image1Data[ getArrayIndex3D(x+1,y+1,z  ,imageDimX,imageDimY) ];
        neighborhood_cache[3].y = d_image2Data[ getArrayIndex3D(x+1,y+1,z  ,imageDimX,imageDimY) ];

        neighborhood_cache[4].x = d_image1Data[ getArrayIndex3D(x  ,y  ,z+1,imageDimX,imageDimY) ];
        neighborhood_cache[4].y = d_image2Data[ getArrayIndex3D(x  ,y  ,z+1,imageDimX,imageDimY) ];

        neighborhood_cache[5].x = d_image1Data[ getArrayIndex3D(x+1,y  ,z+1,imageDimX,imageDimY) ];
        neighborhood_cache[5].y = d_image2Data[ getArrayIndex3D(x+1,y  ,z+1,imageDimX,imageDimY) ];

        neighborhood_cache[6].x = d_image1Data[ getArrayIndex3D(x  ,y+1,z+1,imageDimX,imageDimY) ];
        neighborhood_cache[6].y = d_image2Data[ getArrayIndex3D(x  ,y+1,z+1,imageDimX,imageDimY) ];

        neighborhood_cache[7].x = d_image1Data[ getArrayIndex3D(x+1,y+1,z+1,imageDimX,imageDimY) ];
        neighborhood_cache[7].y = d_image2Data[ getArrayIndex3D(x+1,y+1,z+1,imageDimX,imageDimY) ];

        // transform image intensities to probability space
        for ( unsigned int i = 0; i < 8; ++i )
        {
            neighborhood_cache[i].x = ( neighborhood_cache[i].x - image1Min ) * invBinSize1;
            neighborhood_cache[i].y = ( neighborhood_cache[i].y - image2Min ) * invBinSize2;
        }

        // in the geometry data, get the n-hood index ...
        unsigned int nHoodIdx = getArrayIndex3D( x,y,z, imageDimX-1, imageDimY-1 );
        // ... and where we are actually going to write
        unsigned int geo_unit_offset = 5 * FLOATS_PER_GEO_UNIT * nHoodIdx;

        // extract geometry
        // alpha, beta, gamma, epsilon
        getGeometryFromVertices(
                neighborhood_cache[0], neighborhood_cache[1],
                neighborhood_cache[2], neighborhood_cache[4],
                d_geometryData, geo_unit_offset,
                outerWeight );
        geo_unit_offset += FLOATS_PER_GEO_UNIT;

        // beta, gamma, delta, theta
        getGeometryFromVertices(
                neighborhood_cache[1], neighborhood_cache[2],
                neighborhood_cache[3], neighborhood_cache[7],
                d_geometryData, geo_unit_offset,
                outerWeight );
        geo_unit_offset += FLOATS_PER_GEO_UNIT;

        // beta, epsilon, dzeta, theta
        getGeometryFromVertices(
                neighborhood_cache[1], neighborhood_cache[4],
                neighborhood_cache[5], neighborhood_cache[7],
                d_geometryData, geo_unit_offset,
                outerWeight );
        geo_unit_offset += FLOATS_PER_GEO_UNIT;

        // gamma, epsilon, eta, theta
        getGeometryFromVertices(
                neighborhood_cache[2], neighborhood_cache[4],
                neighborhood_cache[6], neighborhood_cache[7],
                d_geometryData, geo_unit_offset,
                outerWeight );
        geo_unit_offset += FLOATS_PER_GEO_UNIT;

        // beta, gamma, epsilon, dzeta
        getGeometryFromVertices(
                neighborhood_cache[1], neighborhood_cache[2],
                neighborhood_cache[4], neighborhood_cache[5],
                d_geometryData, geo_unit_offset,
                innerWeight );
    }

}

/*
 * if we don't care about the histogram, just the mutual information,
 * the joint histogram doesn't need to leave the gpu.
 * in that case, pass the resultData device pointer to render points on,
 * resulting in a faster execution.
 * Otherwise, pass a null-pointer (default)
 */
bool ExtractGeometryFromImagesCUDA(
        unsigned int xDim,
        unsigned int yDim,
        unsigned int zDim,
        float * d_geometryData,
        float * d_image1Data,
        float * d_image2Data,
        float   image1Min,
        float   image2Min,
        float   binSize1,
        float   binSize2
        )
{
    hipError_t err;

    // every block is 1D, with as many elements as neighborhoods
    // in the z-dimension (of the image)
    dim3 dimBlock( zDim-1, 1     , 1 );
    // the grid is 2D, one block for every x-y neighborhood
    dim3 dimGrid ( xDim-1, yDim-1, 1 );

    //Precomputes the weightings of the inner tetrahedron and four outer ones and
    //the no. nhoods to do this.
    unsigned int  n_neighborhoods = (xDim-1) * (yDim-1) * (zDim-1);
    float         innerWeight     = 1.0f / ( ((float)n_neighborhoods) * 3.0f );
    float         outerWeight     = innerWeight * 0.5f;

    float invBinSize1 = 1.0f / binSize1;
    float invBinSize2 = 1.0f / binSize2;

    ExtractGeometryFromImagesKernel<<<dimGrid, dimBlock>>>(
            d_geometryData,
            d_image1Data,
            d_image2Data,
            xDim, yDim, zDim,
            innerWeight,
            outerWeight,
            image1Min,
            image2Min,
            invBinSize1,
            invBinSize2
            );

    //Ensures we have finished writing into d_geometryData (we call this after every kernel)
    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Executing ExtractGeometryFromImagesKernel: %s\n", hipGetErrorString(err));
        return false;
    }

    return true;
}


