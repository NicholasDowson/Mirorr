#include "hip/hip_runtime.h"
/*
 * NPWCudaRenderGeometryDirectlyFromCUDA.cu
 *
 *  Created on: 28/07/2010
 *      Author: bro86j
 */

#include "NPWCudaConstants.h"
#include "NPWCudaGeometryFunctions.h"
#include "NPWCudaDataPointer.h"
#include "NPWCudaVertexBufferPointer.h"
#include <cudpp.h>
#include <stdio.h>


__global__ void GetTriangleCountKernel(
        int *   d_triangleCount,
        int *   d_vertexCount,
        float * d_geoData,
        int     geoUnitCount)
{
    int geoUnit_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if ( geoUnit_idx < geoUnitCount )
    {

        int nTriangles = (int)d_geoData[ geoUnit_idx * FLOATS_PER_GEO_UNIT ];

        d_triangleCount[geoUnit_idx] = nTriangles;

        int nVertices;

        switch ( nTriangles )
        {
            case 0:
            {
                nVertices = 0;
                break;
            }
            case 1:
            {
                nVertices = 3;
                break;
            }
            case 2:
            {
                nVertices = 4;
                break;
            }
            case 3:
            {
                nVertices = 4;
                break;
            }
            case 4:
            {
                nVertices = 5;
                break;
            }

            default:
            {
                nVertices = 0;
                break;
            }
        }

        d_vertexCount[geoUnit_idx] = nVertices;
    }

}

__global__ void FillBuffersKernel(
        float * d_VertexBuffer,
        int *   d_IndexBuffer,
        int *   d_TriangleOffsets,
        int *   d_VertexOffsets,
        float * d_GeoData,
        int     geoUnitCount,
        float   renderOffset,
        float * d_pointResultData,
        int     resultDataDim
        )
{
    int geoUnit_idx = threadIdx.x + blockIdx.x * blockDim.x;

    if ( geoUnit_idx < geoUnitCount )
    {
        int geo_idx               = FLOATS_PER_GEO_UNIT * geoUnit_idx;
            // where in d_geoData the data for this unit begins
        int vertexBuffer_idx      = d_VertexOffsets[geoUnit_idx];
            // which vertex is the first vertex we are going to write
        int vertexfloatBuffer_idx = vertexBuffer_idx * 3;
            // but we're writing 3 floats per vertex, so multiply by 3
        int indexBuffer_idx       = d_TriangleOffsets[geoUnit_idx] * 3;
            // which triangle index is the first index we are going to write
            //   (multiplied by 3, cause there are 3 vertices per triangle)

        switch ( (int) d_GeoData[geo_idx] )
        {
            case 0:
            {
                int x = (int)d_GeoData[geo_idx+2];
                int y = (int)d_GeoData[geo_idx+3];

                int result_idx = getArrayIndex2D( x,y, resultDataDim,resultDataDim );
                atomicAdd( d_pointResultData + result_idx, d_GeoData[geo_idx+1] );

                break;
            }

            case 1:
            {
                d_VertexBuffer[vertexfloatBuffer_idx    ] = d_GeoData[geo_idx+2]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 1] = d_GeoData[geo_idx+3]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 2] = d_GeoData[geo_idx+1];

                d_VertexBuffer[vertexfloatBuffer_idx + 3] = d_GeoData[geo_idx+4]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 4] = d_GeoData[geo_idx+5]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 5] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 6] = d_GeoData[geo_idx+6]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 7] = d_GeoData[geo_idx+7]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 8] = 0.0f;

                d_IndexBuffer[indexBuffer_idx    ] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 1] = vertexBuffer_idx+1;
                d_IndexBuffer[indexBuffer_idx + 2] = vertexBuffer_idx+2;

                break;
            }
            case 2:
            {
                d_VertexBuffer[vertexfloatBuffer_idx    ] = d_GeoData[geo_idx+2]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 1] = d_GeoData[geo_idx+3]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 2] = d_GeoData[geo_idx+1];

                d_VertexBuffer[vertexfloatBuffer_idx + 3] = d_GeoData[geo_idx+4]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 4] = d_GeoData[geo_idx+5]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 5] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 6] = d_GeoData[geo_idx+6]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 7] = d_GeoData[geo_idx+7]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 8] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 9] = d_GeoData[geo_idx+8]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +10] = d_GeoData[geo_idx+9]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +11] = 0.0f;

                d_IndexBuffer[indexBuffer_idx    ] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 1] = vertexBuffer_idx+1;
                d_IndexBuffer[indexBuffer_idx + 2] = vertexBuffer_idx+2;

                d_IndexBuffer[indexBuffer_idx + 3] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 4] = vertexBuffer_idx+1;
                d_IndexBuffer[indexBuffer_idx + 5] = vertexBuffer_idx+3;

                break;
            }
            case 3:
            {
                d_VertexBuffer[vertexfloatBuffer_idx    ] = d_GeoData[geo_idx+2]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 1] = d_GeoData[geo_idx+3]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 2] = d_GeoData[geo_idx+1];

                d_VertexBuffer[vertexfloatBuffer_idx + 3] = d_GeoData[geo_idx+4]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 4] = d_GeoData[geo_idx+5]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 5] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 6] = d_GeoData[geo_idx+6]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 7] = d_GeoData[geo_idx+7]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 8] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 9] = d_GeoData[geo_idx+8]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +10] = d_GeoData[geo_idx+9]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +11] = 0.0f;

                d_IndexBuffer[indexBuffer_idx    ] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 1] = vertexBuffer_idx+1;
                d_IndexBuffer[indexBuffer_idx + 2] = vertexBuffer_idx+2;

                d_IndexBuffer[indexBuffer_idx + 3] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 4] = vertexBuffer_idx+2;
                d_IndexBuffer[indexBuffer_idx + 5] = vertexBuffer_idx+3;

                d_IndexBuffer[indexBuffer_idx + 6] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 7] = vertexBuffer_idx+3;
                d_IndexBuffer[indexBuffer_idx + 8] = vertexBuffer_idx+1;

                break;
            }
            case 4:
            {
                d_VertexBuffer[vertexfloatBuffer_idx     ] = d_GeoData[geo_idx + 2]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  1] = d_GeoData[geo_idx + 3]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  2] = d_GeoData[geo_idx + 1];

                d_VertexBuffer[vertexfloatBuffer_idx +  3] = d_GeoData[geo_idx + 4]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  4] = d_GeoData[geo_idx + 5]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  5] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx +  6] = d_GeoData[geo_idx + 6]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  7] = d_GeoData[geo_idx + 7]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx +  8] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx +  9] = d_GeoData[geo_idx + 8]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 10] = d_GeoData[geo_idx + 9]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 11] = 0.0f;

                d_VertexBuffer[vertexfloatBuffer_idx + 12] = d_GeoData[geo_idx + 10]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 13] = d_GeoData[geo_idx + 11]+renderOffset;
                d_VertexBuffer[vertexfloatBuffer_idx + 14] = 0.0f;

                d_IndexBuffer[indexBuffer_idx     ] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx +  1] = vertexBuffer_idx + 1;
                d_IndexBuffer[indexBuffer_idx +  2] = vertexBuffer_idx + 2;

                d_IndexBuffer[indexBuffer_idx +  3] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx +  4] = vertexBuffer_idx + 2;
                d_IndexBuffer[indexBuffer_idx +  5] = vertexBuffer_idx + 3;

                d_IndexBuffer[indexBuffer_idx +  6] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx +  7] = vertexBuffer_idx + 3;
                d_IndexBuffer[indexBuffer_idx +  8] = vertexBuffer_idx + 4;

                d_IndexBuffer[indexBuffer_idx +  9] = vertexBuffer_idx;
                d_IndexBuffer[indexBuffer_idx + 10] = vertexBuffer_idx + 4;
                d_IndexBuffer[indexBuffer_idx + 11] = vertexBuffer_idx + 1;

                break;
            }

            default:
            {
                break;
            }
        }

    }
}

bool RenderGeometryDirectlyFromCUDA(
        float *       d_GeoData,
        int           geoDataSize,
        float         renderOffset,
        float *       d_pointResultData,
        int           resultDataDim
        )
{
    /*
     * create cuda array to store #tri's each shared data
     */
    int geoUnitCount = geoDataSize / FLOATS_PER_GEO_UNIT;

    NPWCudaDataPointer<int> * triangleCount =
            new NPWCudaDataPointer<int>( geoUnitCount , 0, true,true );

    NPWCudaDataPointer<int> * vertexCount =
            new NPWCudaDataPointer<int>( geoUnitCount , 0, true,true );

    dim3 dimBlock(128,1,1);
    dim3 dimGrid( (int)ceil((float)geoUnitCount/128.0f), 1, 1);

    GetTriangleCountKernel<<<dimGrid, dimBlock>>>(
            triangleCount->GetRawCudaPointer(),
            vertexCount->GetRawCudaPointer(),
            d_GeoData,
            geoUnitCount );

    hipDeviceSynchronize();

    hipError_t err;

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Executing getTriangleCountKernel: %s\n", hipGetErrorString(err));
        return false;
    }

    /*
     * get offsets for index- and vertexbuffer
     */
    NPWCudaDataPointer<int> * triangleOffsets =
                new NPWCudaDataPointer<int>( geoUnitCount, 0, true,true );

    NPWCudaDataPointer<int> * vertexOffsets =
                new NPWCudaDataPointer<int>( geoUnitCount, 0, true,true );

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_INT;
    config.algorithm = CUDPP_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_EXCLUSIVE;

    CUDPPHandle scanplan = 0;
    CUDPPResult result = cudppPlan(&scanplan, config, geoUnitCount, 1, 0);


    if (CUDPP_SUCCESS != result)
    {
        printf("Error creating CUDPPPlan\n");
        exit(-1);
    }

    /*
     * get total number of triangles
     */
    cudppScan( scanplan,
               triangleOffsets->GetRawCudaPointer(),
               triangleCount->GetRawCudaPointer(),
               geoUnitCount );

    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Executing Prefix Sum Kernel cudppScan to get triangle offsets: %s\n", hipGetErrorString(err));
        return false;
    }

    int lastTriangleOffset;
    hipMemcpy( &lastTriangleOffset,
                triangleOffsets->GetRawCudaPointer() + geoUnitCount - 1 ,
                sizeof(int),
                hipMemcpyDeviceToHost );

    int lastTriangleCount;
    hipMemcpy( &lastTriangleCount,
                triangleCount->GetRawCudaPointer() + geoUnitCount - 1 ,
                sizeof(int),
                hipMemcpyDeviceToHost );

    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Copying triangle count from device: %s\n", hipGetErrorString(err));
        return false;
    }

    int nAllTriangles = lastTriangleOffset + lastTriangleCount;

    /*
     * get total number of vertices
     */
    cudppScan( scanplan,
               vertexOffsets->GetRawCudaPointer(),
               vertexCount->GetRawCudaPointer(),
               geoUnitCount );

    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Executing Prefix Sum Kernel cudppScan to get vertex offsets: %s\n", hipGetErrorString(err));
        return false;
    }

    result = cudppDestroyPlan(scanplan);
    if (CUDPP_SUCCESS != result)
    {
        printf("Error destroying CUDPPPlan\n");
        exit(-1);
    }

    int lastVertexOffset = 0;
    hipMemcpy( &lastVertexOffset,
                vertexOffsets->GetRawCudaPointer() + geoUnitCount - 1 ,
                sizeof(int),
                hipMemcpyDeviceToHost );

    int lastVertexCount = 0;
    hipMemcpy( &lastVertexCount,
                vertexCount->GetRawCudaPointer() + geoUnitCount - 1 ,
                sizeof(int),
                hipMemcpyDeviceToHost );

    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Copying vertex count from device: %s\n", hipGetErrorString(err));
        return false;
    }

    int nAllVertices = lastVertexOffset + lastVertexCount;

    /*
     * create buffers
     */
    NPWCudaVertexBufferPointer<int> *   triangleIndexBuffer  = 0;
    NPWCudaVertexBufferPointer<float> * triangleVertexBuffer = 0;

    int *   d_triangleIndexBuffer = 0;
    float * d_vertexIndexBuffer   = 0;

    if ( nAllTriangles != 0 )
    {
        // 3 vertices/triangle
        triangleIndexBuffer =
                    new NPWCudaVertexBufferPointer<int>( nAllTriangles * 3 );
        d_triangleIndexBuffer = triangleIndexBuffer->GetBufferAsCudaPointer();
        // 3 floats/vertex
        triangleVertexBuffer =
                    new NPWCudaVertexBufferPointer<float>( nAllVertices * 3 );
        d_vertexIndexBuffer = triangleVertexBuffer->GetBufferAsCudaPointer();

        if ( !d_triangleIndexBuffer || !d_vertexIndexBuffer )
        {
            std::cout << "RenderGeometryDirectlyFromCUDA(): vertex buffer null pointer!" << std::endl;
            if (triangleIndexBuffer)    { triangleIndexBuffer->ReleaseBuffer(); }
            if (triangleVertexBuffer)   { triangleVertexBuffer->ReleaseBuffer(); }

            if (triangleCount)          { delete triangleCount;         triangleCount   = 0; }
            if (vertexCount)            { delete vertexCount;           vertexCount     = 0; }
            if (triangleOffsets)        { delete triangleOffsets;       triangleOffsets = 0; }
            if (vertexOffsets)          { delete vertexOffsets;         vertexOffsets   = 0; }
            if (triangleIndexBuffer)    { delete triangleIndexBuffer;   triangleIndexBuffer  = 0; }
            if (triangleVertexBuffer)   { delete triangleVertexBuffer;  triangleVertexBuffer = 0; }

            std::cout << "RenderGeometryDirectlyFromCUDA() cleaned up stuff" << std::endl;
            return false;
        }
    }

    FillBuffersKernel<<<dimGrid, dimBlock>>>(
            d_vertexIndexBuffer,
            d_triangleIndexBuffer,
            triangleOffsets->GetRawCudaPointer(),
            vertexOffsets->GetRawCudaPointer(),
            d_GeoData,
            geoUnitCount,
            renderOffset,
            d_pointResultData,
            resultDataDim
            );

    hipDeviceSynchronize();

    if (((err = hipGetLastError())) != hipSuccess)
    {
        printf("Error Executing FillBuffersKernel: %s\n", hipGetErrorString(err));
        return false;
    }

    if ( triangleIndexBuffer && triangleVertexBuffer )
    {
        triangleIndexBuffer->ReleaseBuffer();
        triangleVertexBuffer->ReleaseBuffer();

        glPushClientAttrib(GL_CLIENT_ALL_ATTRIB_BITS);

        // bind the vertex buffer
        glBindBuffer( GL_ARRAY_BUFFER, triangleVertexBuffer->GetVertexBufferID() );
        glVertexPointer( 3, GL_FLOAT, 3 * sizeof(float), (GLubyte*) (0));
        glEnableClientState(GL_VERTEX_ARRAY);

        // render all the triangles
        glBindBuffer( GL_ELEMENT_ARRAY_BUFFER, triangleIndexBuffer->GetVertexBufferID() );
        glDrawElements( GL_TRIANGLES, 3 * nAllTriangles, GL_UNSIGNED_INT, 0);
        glBindBuffer(GL_ELEMENT_ARRAY_BUFFER,0);
        glBindBuffer(GL_ARRAY_BUFFER,0);
        glPopClientAttrib();

        glFinish();

        CheckOpenGLError();
    }

    /*
     * clean up
     */
    if (triangleCount)          { delete triangleCount;        triangleCount        = 0; }
    if (vertexCount)            { delete vertexCount;          vertexCount          = 0; }
    if (triangleOffsets)        { delete triangleOffsets;      triangleOffsets      = 0; }
    if (vertexOffsets)          { delete vertexOffsets;        vertexOffsets        = 0; }
    if (triangleIndexBuffer)    { delete triangleIndexBuffer;  triangleIndexBuffer  = 0; }
    if (triangleVertexBuffer)   { delete triangleVertexBuffer; triangleVertexBuffer = 0; }

    // done!
    return true;
}
